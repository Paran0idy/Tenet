#include <hip/hip_runtime.h>
#include <iostream>

// block tile
#define BLOCK_M 128
#define BLOCK_N 128
#define BLOCK_K 8

// element per thread  
#define THREAD_N 8

#define OFFSET(i, j, N) ((i) * (N) + (j)) 
#define FLOAT4(pointer) reinterpret_cast<float4*>(&pointer)[0]
__global__ void matmul(float *a, float *b, float *c, int M, int N, int K){

    __shared__ float shared_a[BLOCK_M][BLOCK_K];
    __shared__ float shared_b[BLOCK_K][BLOCK_N];

    float res[THREAD_N][THREAD_N] = {0};

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int smem_a_m = tid / 2;
    int smem_a_k = (tid % 2) * 4;

    int smem_b_k = tid / 32;
    int smem_b_n = (tid % 32) * 4;

    int gmem_a_m = BLOCK_M * blockIdx.y + smem_a_m;
    int gmem_b_n = BLOCK_N * blockIdx.x + smem_b_n;

    for(int k = 0; k < K / BLOCK_K; k++){
        // GMEM -> SMEM
        int gmem_a_k = k * BLOCK_K + smem_a_k;
        int gmem_b_k = k * BLOCK_K + smem_b_k;

        FLOAT4(shared_a[smem_a_m][smem_a_k]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        FLOAT4(shared_b[smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);
        __syncthreads();

        // Compute
        int ty = threadIdx.y * THREAD_N;
        int tx = threadIdx.x * THREAD_N;
        for(int kk = 0; kk < BLOCK_K; kk++)
            for(int ii = 0; ii < THREAD_N; ii++)
                for(int jj = 0; jj < THREAD_N; jj++)
                    res[ii][jj] += shared_a[ty + ii][kk] * shared_b[kk][tx + jj];
        __syncthreads();
    }

    // Write back
    int ty = BLOCK_M * blockIdx.y + THREAD_N * threadIdx.y;
    int tx = BLOCK_N * blockIdx.x + THREAD_N * threadIdx.x;

    for(int i = 0; i < THREAD_N; i++)
        for(int j = 0; j < THREAD_N; j++)
            c[OFFSET(ty + i, tx + j, N)] = res[i][j];
}


int main(){
    // Problem size
    int M = 128;
    int N = 128;
    int K = 128;
    // Host
    float *a = (float *)malloc(M * K * sizeof(float));
    float *b = (float *)malloc(K * N * sizeof(float));
    float *c = (float *)malloc(M * N * sizeof(float));
    for(int i = 0; i < M * K; i++) 
        a[i] = 1;
    for(int i = 0; i < K * N; i++)
        b[i] = 1;
    
    // Device
    float *da, *db, *dc;
    hipMalloc(&da, M * K * sizeof(float));
    hipMalloc(&db, K * N * sizeof(float));
    hipMalloc(&dc, M * N * sizeof(float));

    // Copy to device
    hipMemcpy(da, a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, K * N * sizeof(float), hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N / THREAD_N, BLOCK_M / THREAD_N);
    matmul<<<grid, block>>>(da, db, dc, M, N, K);

    // Copy to host
    hipMemcpy(c, dc, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++)
            std::cout << c[i * N + j] << " ";
        std::cout << std::endl;
    }

    // Free
    free(a);
    free(b);
    free(c);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}